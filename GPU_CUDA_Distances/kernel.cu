
#include "hip/hip_runtime.h"



#include <stdio.h>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//rows determined as the amount of rows in a block
// A is query vector, B is the model ( rows ), C is output matrix
// Rows should be 300 for proper usage of this access method
__global__ void DotProduct
(int rows, float *A, float *B, float *C, float normA, float *normsB) {
  __shared__ float fastA[300];
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id<300) {
      fastA[id]=A[id];
  }
  __syncthreads();
  float acum=0;
  for(int i=0;i<300;++i) {
      acum+=fastA[i]*B[id*300+i];
  }
  C[id]=acum/(normA*normsB[id]);
}


__global__ void FirstMerge
(int N, float *sims, int length) {

  
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int start=id*N;
    int end=start+N;
    if (!(start>length)) { 
    
    // Insertion sort, as N SHOULD be small
   int key, j;
   for(int i = start+1; i<end; i++) {
      key = sims[i];
      j = i;
      while(j > 0 && sims[j-1]<key) {
         sims[j] = sims[j-1];
         j--;
      }
      sims[j] = key;  
   }
}
}



extern "C"
int runCuda()
{
    const int arraySize = 10;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    float sims[arraySize] = { 10, 20, 30, 40, 50,1,5,6,38,123};
    //int c[arraySize] = { 0 };
    
    
    int end=arraySize;
    int start=0;
   int key, j;
   for(int i = start+1; i<end; i++) {
      key = sims[i];
      j = i;
      while(j > 0 && sims[j-1]<key) {
         sims[j] = sims[j-1];
         j--;
      }
      sims[j] = key;  
   }
    for (int i=0;i<arraySize;++i){
        printf("{%f} ",
        sims[i]);

    }
    return 0;

/*
    // Add vectors in parallel.
    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = cudaDeviceReset();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceReset failed!");
        return 1;
    }
	
    return 0;
    */
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}